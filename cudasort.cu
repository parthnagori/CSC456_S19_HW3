
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <sys/time.h>

#define THREADS 512
#ifdef __cplusplus
extern "C"
{
#endif


__global__ void bitonic_sort(float *arr, int k, int j)
{
  unsigned int i, ij; 
    i = threadIdx.x + blockDim.x * blockIdx.x;
  ij = i^j;

  if ((ij)>i) {
    if ((i&k)==0) {
      if (arr[i]>arr[ij]) {
        float temp = arr[i];
        arr[i] = arr[ij];
        arr[ij] = temp;
      }
    }
    if ((i&k)!=0) {
      if (arr[i]<arr[ij]) {
        float temp = arr[i];
        arr[i] = arr[ij];
        arr[ij] = temp;
      }
    }
  }
}


int cuda_sort(int number_of_elements, float *a)
{
  
  float *arr;
  
  hipMalloc((void**) &arr, number_of_elements * sizeof(float));
  hipMemcpy(arr, a, number_of_elements * sizeof(float), hipMemcpyHostToDevice);
  
  dim3 dimGrid(number_of_elements/512,1);
  dim3 dimBlock(512,1);

  for (int i = 2; i <= number_of_elements; i <<= 1) {
    for (int j= i>>1 ; j>0; j = j>>1) {
      bitonic_sort<<<dimGrid, dimBlock>>>(arr, i, j);
    }
  }
  hipMemcpy(a, arr, number_of_elements * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(arr);

  return 0;
}

#ifdef __cplusplus
}
#endif

