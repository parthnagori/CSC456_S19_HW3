#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <sys/time.h>

#define THREADS 512
#ifdef __cplusplus
extern "C"
{
#endif

__device__ void merge(float* arr, float* final, int start, int mid, int end)
{
    int i = start;
    int j = mid;
    for (int k = start; k < end; k++){
      if (i==mid){
        final[k] = arr[j++];
      }
      else if (j == end){
        final[k] = arr[i++];
      }
      else if (arr[i] < arr[j]){
        final[k] = arr[i++];
      }
      else{
        final[k] = arr[j++];
      }
    }
}

__global__ void merge_sort(float* arr, float* final, int numberOfBlocks, int elementsPerBlock, int partition){

    int block_id = blockIdx.x;   
    int start = block_id * partition;
    int end = start + partition;
    int mid = start + partition/2;

    merge(arr, final, start, mid, end);
}

int cuda_sort(int number_of_elements, float *a)
{
  
  float *arr;
  float *final;
  int n;
  int part = 0;

  int numberOfBlocks = 512;
  int elementsPerBlock = number_of_elements/numberOfBlocks;

  hipEvent_t event;
  hipEventCreate(&event);

  hipMalloc((void **) &arr, sizeof(float)*number_of_elements);
  hipMalloc((void **) &final, sizeof(float)*number_of_elements);
  hipMemcpy(arr, a, sizeof(float)*number_of_elements, hipMemcpyHostToDevice);

  dim3 dimGrid(numberOfBlocks);
  dim3 dimBlock(1);

  int partition;
  // int partition_size;

  int cnt = 0;
  for (partition = 2; partition < 2*number_of_elements; partition*=2) {
    if ((cnt % 2) == 0)
      merge_sort<<<dimGrid, dimBlock>>>(arr, final, numberOfBlocks, elementsPerBlock, partition); 
    else
      merge_sort<<<dimGrid, dimBlock>>>(final, arr, numberOfBlocks, elementsPerBlock, partition);
    cnt+=1; 
  }

  hipMemcpy(a, final, sizeof(float)*number_of_elements, hipMemcpyDeviceToHost);
  // hipFree(gpu_arr);
  hipDeviceSynchronize();
  hipEventSynchronize(event);
  
  return 0;
}

#ifdef __cplusplus
}
#endif
