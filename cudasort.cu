
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <sys/time.h>
#define THREADS 512
#ifdef __cplusplus
extern "C"
{
#endif


__global__ void bitonic_sort(float *arr, int i, int j)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int p = index ^ j;
  int q = index & i;
  float temp;
  if (p > index) {    
    if (((q==0) && (arr[index]>arr[p])) || ((q!=0) && (arr[index]<arr[p]))) {
      temp = arr[index];
      arr[index] = arr[p];
      arr[p] = temp;
    }
  }
}


int cuda_sort(int number_of_elements, float *a)
{
  
  float *arr;
  
  hipMalloc((void**) &arr, number_of_elements * sizeof(float));
  hipMemcpy(arr, a, number_of_elements * sizeof(float), hipMemcpyHostToDevice);
  
  dim3 dimGrid(number_of_elements/THREADS);
  dim3 dimBlock(THREADS);
  
  int i = 2;
  while (i <= number_of_elements) {
    int j = i/2;
    while (j > 0){
      bitonic_sort<<<dimGrid, dimBlock>>>(arr, i, j);
      j/=2;
    }
    i*=2;
  }
  hipMemcpy(a, arr, number_of_elements * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(arr);

  return 0;
}

#ifdef __cplusplus
}
#endif

