
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <sys/time.h>

#define THREADS 512
#ifdef __cplusplus
extern "C"
{
#endif


__global__ void bitonic_sort_step(float *arr, int j, int k)
{
  unsigned int i, ij; 
    i = threadIdx.x + blockDim.x * blockIdx.x;
  ij = i^j;

  if ((ij)>i) {
    if ((i&k)==0) {
      if (arr[i]>arr[ij]) {
        float temp = arr[i];
        arr[i] = arr[ij];
        arr[ij] = temp;
      }
    }
    if ((i&k)!=0) {
      if (arr[i]<arr[ij]) {
        float temp = arr[i];
        arr[i] = arr[ij];
        arr[ij] = temp;
      }
    }
  }
}


int cuda_sort(int number_of_elements, float *a)
{
  
  float *arr;
  
  hipMalloc((void**) &arr, number_of_elements * sizeof(float));
  hipMemcpy(arr, a, number_of_elements * sizeof(float), hipMemcpyHostToDevice);
  
  int thread_cnt = 0;
  int block_cnt = 0;
   if(number_of_elements % 512 == 0)
  {
    thread_cnt = 512;
    block_cnt = number_of_elements/512;
  }
  //   else if(number_of_elements < 512){
  //   thread_cnt =number_of_elements;
  //   block_cnt = 1;
  // }
  //   else{
  // thread_cnt = number_of_elements%512;
  // block_cnt = number_of_elements/512;
  // }
  dim3 blocks_per_grid(block_cnt,1);    /* Number of blocks   */
  dim3 threads_per_block(thread_cnt,1);  /* Number of threads  */

  int l, m;
  for (l = 2; l <= number_of_elements; l <<= 1) {
    for (m=l>>1; m>0; m=m>>1) {
      bitonic_sort_step<<<blocks_per_grid, threads_per_block>>>(arr, m, l);
    }
  }
  hipMemcpy(a, arr, number_of_elements * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(arr);

  return 0;
}

#ifdef __cplusplus
}
#endif

