
#include <hip/hip_runtime.h>
 #ifdef __cplusplus
  extern "C"
  {
  #endif


__global__ void bitonic_sort(float *arr, int j, int k)
{
  unsigned int i, ij; 
    i = threadIdx.x + blockDim.x * blockIdx.x;
  ij = i^j;

  if ((ij)>i) {
    if ((i&k)==0) {
      if (arr[i]>arr[ij]) {
        float temp = arr[i];
        arr[i] = arr[ij];
        arr[ij] = temp;
      }
    }
    if ((i&k)!=0) {
      if (arr[i]<arr[ij]) {
        float temp = arr[i];
        arr[i] = arr[ij];
        arr[ij] = temp;
      }
    }
  }
}


int cuda_sort(int number_of_elements, float *a)
{
  
  float *arr;
  
  hipMalloc((void **) &arr, sizeof(float)*number_of_elements);
  hipMemcpy(arr, a, sizeof(float)*number_of_elements, hipMemcpyHostToDevice);

  int threads_create = 0;
  int blocks_create = 0;
   if(number_of_elements % 512 == 0)
  {
    threads_create = 512;
    blocks_create = number_of_elements/512;
  }
    else if(number_of_elements < 512){
    threads_create =number_of_elements;
    blocks_create = 1;
  }
    else{
  threads_create = number_of_elements%512;
  blocks_create = number_of_elements/512;
  }
  // printf("threads : %d, blocks: %d", threads_create, blocks_create);
  dim3 blocks(blocks_create,1);    /* Number of blocks   */
  dim3 threads(threads_create,1);  /* Number of threads  */

  int l, m;
  for (l = 2; l <= number_of_elements; l <<= 1) {
    for (m=l>>1; m>0; m=m>>1) {
      bitonic_sort<<<blocks, threads>>>(arr, m, l);
    }
  }
  hipMemcpy(a, arr, sizeof(float)*number_of_elements, hipMemcpyHostToDevice);
  hipFree(arr);

  return 0;
}

#ifdef __cplusplus
}
#endif

