
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <sys/time.h>
#include <bits/stdc++.h> 
#define THREADS 512
#ifdef __cplusplus
extern "C"
{
#endif


__global__ void bitonic_sort(float *arr, int k, int j)
{
  int ij; 
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  ij = index^j;

  if ((ij)>index) {
    if ((((index & k)==0) && (arr[index]>arr[ij])) || (((index & k)!=0) && (arr[index]<arr[ij]))) {
        float temp = arr[index];
        arr[index] = arr[ij];
        arr[ij] = temp;
    }
  }
}


int cuda_sort(int number_of_elements, float *a)
{
  
  float *arr;
  
  hipMalloc((void**) &arr, number_of_elements * sizeof(float));
  hipMemcpy(arr, a, number_of_elements * sizeof(float), hipMemcpyHostToDevice);
  
  dim3 dimGrid(number_of_elements/512,1);
  dim3 dimBlock(512,1);

  for (int i = 2; i <= number_of_elements; i*=2) {
    int j = i/2;
    while (j > 0){
      bitonic_sort<<<dimGrid, dimBlock>>>(arr, i, j);
      j/=2;
    }
  }
  hipMemcpy(a, arr, number_of_elements * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(arr);

  return 0;
}

#ifdef __cplusplus
}
#endif

