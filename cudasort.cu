#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <sys/time.h>

#define THREADS 512
#ifdef __cplusplus
extern "C"
{
#endif


__global__ void bitonic_sort(float *arr, int j, int k)
{
  unsigned int i, ij; 
    i = threadIdx.x + blockDim.x * blockIdx.x;
  ij = i^j;

  if ((ij)>i) {
    if ((i&k)==0) {
      if (arr[i]>arr[ij]) {
        float temp = arr[i];
        arr[i] = arr[ij];
        arr[ij] = temp;
      }
    }
    if ((i&k)!=0) {
      if (arr[i]<arr[ij]) {
        float temp = arr[i];
        arr[i] = arr[ij];
        arr[ij] = temp;
      }
    }
  }
}


int cuda_sort(int number_of_elements, float *a)
{
  
  float *arr;
  
  hipEvent_t event;
  hipEventCreate(&event);

  hipMalloc((void**) &arr, number_of_elements * sizeof(float));
  hipMemcpy(arr, a, number_of_elements * sizeof(float), hipMemcpyHostToDevice);
  
  
  dim3 dimGrid(512,1);    
  dim3 dimBlock(number_of_elements/512,1);

  int l, m;
  for (l = 2; l <= number_of_elements; l <<= 1) {
    for (m=l>>1; m>0; m=m>>1) {
      bitonic_sort<<<dimGrid, dimBlock>>>(arr, m, l);
    }
  }

  hipMemcpy(a, arr, number_of_elements * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(arr);
  hipDeviceSynchronize();
  hipEventSynchronize(event);

  return 0;
}

#ifdef __cplusplus
}
#endif

