#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <sys/time.h>

#define THREADS 512
#ifdef __cplusplus
extern "C"
{
#endif

__device__ void merge(float* arr, float* final, int start, int mid, int end)
{
    int i = start;
    int j = mid;
    int k = start;
    printf("start : %d mid: %d end: %d", start, mid, end);
    while (k < end)
    {
      if (i==mid){
        final[k] = arr[j++];
      }
      else if (j == end){
        final[k] = arr[i++];
      }
      else if (arr[i] < arr[j]){
        final[k] = arr[i++];
      }
      else{
        final[k] = arr[j++];
      }
      k++;
    }

    // for(i = start; i < end; i++)
    // {
    //   arr[i] = final[i];
    // }

}

__global__ void merge_sort(float* arr, float* final, int numberOfBlocks, int elementsPerBlock, int partition){

    int block_id = blockIdx.x;
    int start = block_id * partition;
    int n = numberOfBlocks*elementsPerBlock;
    int end = min(start + partition,n);
    int mid = min(start + partition/2,n);
    merge(arr, final, start, mid, end);
}

int cuda_sort(int number_of_elements, float *a)
{
  
  float *arr;
  float *final;
  // int n;
  // int part = 0;

  int numberOfBlocks = 512;
  int elementsPerBlock = number_of_elements/numberOfBlocks;

  hipEvent_t event;
  hipEventCreate(&event);

  hipMalloc((void **) &arr, sizeof(float)*number_of_elements);
  hipMalloc((void **) &final, sizeof(float)*number_of_elements);
  hipMemcpy(arr, a, sizeof(float)*number_of_elements, hipMemcpyHostToDevice);

  dim3 dimGrid(numberOfBlocks);
  dim3 dimBlock(1);

  int partition;
  // int partitions;

  int cnt = 0;
  // n = number_of_elements;
  // while (n != 0){
  //   ++partitions;
  //   n/=2;
  // } 


  // for (part = 0; part < partitions; part++){
  //   int part_size = part << 1;
  //   merge_sort<<<dimGrid, dimBlock>>>(arr, final, numberOfBlocks, elementsPerBlock, part); 
  // } 


  for (partition = 2; partition < 2*number_of_elements; partition*=2) {
    if (cnt%2 == 0)
      merge_sort<<<dimGrid, dimBlock>>>(arr, final, numberOfBlocks, elementsPerBlock, partition); 
    else
      merge_sort<<<dimGrid, dimBlock>>>(final, arr, numberOfBlocks, elementsPerBlock, partition);
    cnt+=1; 
  }

  printf("cnt: %d", cnt);
  hipMemcpy(a, arr, sizeof(float)*number_of_elements, hipMemcpyDeviceToHost);
  // hipFree(gpu_arr);
  hipDeviceSynchronize();
  hipEventSynchronize(event);
  
  return 0;
}

#ifdef __cplusplus
}
#endif
