
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <sys/time.h>
#define THREADS 512
#ifdef __cplusplus
extern "C"
{
#endif


__global__ void bitonic_sort(float *arr, int i, int j)
{
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  int k = index^j;
  float temp;
  if ((k) > index) {
    if ((((index & i)==0) && (arr[index]>arr[k])) || (((index & i)!=0) && (arr[index]<arr[k]))) {
        temp = arr[index];
        arr[index] = arr[k];
        arr[k] = temp;
    }
  }
}


int cuda_sort(int number_of_elements, float *a)
{
  
  float *arr;
  
  hipMalloc((void**) &arr, number_of_elements * sizeof(float));
  hipMemcpy(arr, a, number_of_elements * sizeof(float), hipMemcpyHostToDevice);
  
  dim3 dimGrid(number_of_elements/512,1);
  dim3 dimBlock(512,1);

  for (int i = 2; i <= number_of_elements; i*=2) {
    int j = i/2;
    while (j > 0){
      bitonic_sort<<<dimGrid, dimBlock>>>(arr, i, j);
      j/=2;
    }
  }
  hipMemcpy(a, arr, number_of_elements * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(arr);

  return 0;
}

#ifdef __cplusplus
}
#endif

