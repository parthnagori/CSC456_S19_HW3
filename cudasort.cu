
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <sys/time.h>
#define THREADS 512
#ifdef __cplusplus
extern "C"
{
#endif


__global__ void bitonic_sort(float *arr, int i, int j)
{
  int index = blockIdx.x;
  int k = index ^ j;
  int l = index & i;
  float temp;
  if (k > index) {    
    if (((l==0) && (arr[index]>arr[k])) || ((l!=0) && (arr[index]<arr[k]))) {
      temp = arr[index];
      arr[index] = arr[k];
      arr[k] = temp;
    }
  }
}


int cuda_sort(int number_of_elements, float *a)
{
  
  float *arr;
  
  hipMalloc((void**) &arr, number_of_elements * sizeof(float));
  hipMemcpy(arr, a, number_of_elements * sizeof(float), hipMemcpyHostToDevice);
  
  dim3 dimGrid(number_of_elements/THREADS);
  dim3 dimBlock(1);
  
  int i = 2;
  while (i <= number_of_elements) {
    int j = i/2;
    while (j > 0){
      bitonic_sort<<<dimGrid, dimBlock>>>(arr, i, j);
      j/=2;
    }
    i*=2;
  }
  hipMemcpy(a, arr, number_of_elements * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(arr);

  return 0;
}

#ifdef __cplusplus
}
#endif

