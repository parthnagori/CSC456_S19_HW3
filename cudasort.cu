
#include <hip/hip_runtime.h>
 #ifdef __cplusplus
  extern "C"
  {
  #endif


__global__ void bitonic_sort_step(float *gpu_val, int j, int k)
{
  unsigned int i, ij; 
    i = threadIdx.x + blockDim.x * blockIdx.x;
  ij = i^j;

  if ((ij)>i) {
    if ((i&k)==0) {
      if (gpu_val[i]>gpu_val[ij]) {
        float temp = gpu_val[i];
        gpu_val[i] = gpu_val[ij];
        gpu_val[ij] = temp;
      }
    }
    if ((i&k)!=0) {
      if (gpu_val[i]<gpu_val[ij]) {
        float temp = gpu_val[i];
        gpu_val[i] = gpu_val[ij];
        gpu_val[ij] = temp;
      }
    }
  }
}


int cuda_sort(int number_of_elements, float *values)
{
  
  float *gpu_arr;
  size_t size = number_of_elements * sizeof(float);

  hipMalloc((void**) &gpu_arr, size);
  hipMemcpy(gpu_arr, values, size, hipMemcpyHostToDevice);
  
  int threads_create = 0;
  int blocks_create = 0;
   if(number_of_elements % 512 == 0)
  {
    threads_create = 512;
    blocks_create = number_of_elements/512;
  }
    else if(number_of_elements < 512){
    threads_create =number_of_elements;
    blocks_create = 1;
  }
    else{
  threads_create = number_of_elements%512;
  blocks_create = number_of_elements/512;
  }
  dim3 blocks(blocks_create,1);    /* Number of blocks   */
  dim3 threads(threads_create,1);  /* Number of threads  */

  int l, m;
  for (l = 2; l <= number_of_elements; l <<= 1) {
    for (m=l>>1; m>0; m=m>>1) {
      bitonic_sort_step<<<blocks, threads>>>(gpu_arr, m, l);
    }
  }
  hipMemcpy(values, gpu_arr, size, hipMemcpyDeviceToHost);
  hipFree(gpu_arr);

  return 0;
}

#ifdef __cplusplus
}
#endif

