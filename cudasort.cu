
#include <hip/hip_runtime.h>
 #ifdef __cplusplus
  extern "C"
  {
  #endif


__global__ void bitonic_sort(float *gpu_val, int j, int k)
{
  unsigned int i, ij; 
    i = threadIdx.x + blockDim.x * blockIdx.x;
  ij = i^j;

  if ((ij)>i) {
    if ((i&k)==0) {
      if (gpu_val[i]>gpu_val[ij]) {
        float temp = gpu_val[i];
        gpu_val[i] = gpu_val[ij];
        gpu_val[ij] = temp;
      }
    }
    if ((i&k)!=0) {
      if (gpu_val[i]<gpu_val[ij]) {
        float temp = gpu_val[i];
        gpu_val[i] = gpu_val[ij];
        gpu_val[ij] = temp;
      }
    }
  }
}


int cuda_sort(int number_of_elements, float *a)
{
  
  float *arr;
  
  hipMalloc((void **) &arr, sizeof(float)*number_of_elements);
  hipMemcpy(arr, a, sizeof(float)*number_of_elements, hipMemcpyHostToDevice);

  int threads_create = 512;
  int blocks_create = number_of_elements/512;
  //  if(number_of_elements % 512 == 0)
  // {
  //   threads_create = 512;
  //   blocks_create = number_of_elements/512;
  // }
  //   else if(number_of_elements < 512){
  //   threads_create =number_of_elements;
  //   blocks_create = 1;
  // }
  //   else{
  // threads_create = number_of_elements%512;
  // blocks_create = number_of_elements/512;
  // }
  dim3 blocks(blocks_create,1);    /* Number of blocks   */
  dim3 threads(threads_create,1);  /* Number of threads  */

  int l, m;
  for (l = 2; l <= number_of_elements; l <<= 1) {
    for (m=l>>1; m>0; m=m>>1) {
      bitonic_sort<<<blocks, threads>>>(arr, m, l);
    }
  }
  hipMemcpy(a, arr, sizeof(float)*number_of_elements, hipMemcpyHostToDevice);
  hipFree(arr);

  return 0;
}

#ifdef __cplusplus
}
#endif

