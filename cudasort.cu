
#include <hip/hip_runtime.h>
 #ifdef __cplusplus
  extern "C"
  {
  #endif


__global__ void bitonic_sort_step(float *arr, int j, int k)
{
  unsigned int i, ij; 
    i = threadIdx.x + blockDim.x * blockIdx.x;
  ij = i^j;

  if ((ij)>i) {
    if ((i&k)==0) {
      if (arr[i]>arr[ij]) {
        float temp = arr[i];
        arr[i] = arr[ij];
        arr[ij] = temp;
      }
    }
    if ((i&k)!=0) {
      if (arr[i]<arr[ij]) {
        float temp = arr[i];
        arr[i] = arr[ij];
        arr[ij] = temp;
      }
    }
  }
}


int cuda_sort(int number_of_elements, float *a)
{
  
  float *arr;
  size_t size = number_of_elements * sizeof(float);

  hipMalloc((void**) &arr, number_of_elements * sizeof(float));
  hipMemcpy(arr, a, number_of_elements * sizeof(float), hipMemcpyHostToDevice);
  
  int threads_create = 0;
  int blocks_create = 0;
   if(number_of_elements % 512 == 0)
  {
    threads_create = 512;
    blocks_create = number_of_elements/512;
  }
    else if(number_of_elements < 512){
    threads_create =number_of_elements;
    blocks_create = 1;
  }
    else{
  threads_create = number_of_elements%512;
  blocks_create = number_of_elements/512;
  }
  dim3 blocks(blocks_create,1);    /* Number of blocks   */
  dim3 threads(threads_create,1);  /* Number of threads  */

  int l, m;
  for (l = 2; l <= number_of_elements; l <<= 1) {
    for (m=l>>1; m>0; m=m>>1) {
      bitonic_sort_step<<<blocks, threads>>>(arr, m, l);
    }
  }
  hipMemcpy(a, arr, number_of_elements * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(arr);

  return 0;
}

#ifdef __cplusplus
}
#endif

