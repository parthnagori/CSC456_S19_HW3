
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <sys/time.h>
#define THREADS 512
#ifdef __cplusplus
extern "C"
{
#endif


__global__ void bitonic_sort(float *arr, int i, int j)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  // printf("\nidx : %d dim: %d th: %d",blockIdx.x, blockDim.x, threadIdx.x);
  int k = index ^ j;
  int l = index & i;
  float temp;
  if (k > index) {    
    if (((l==0) && (arr[index]>arr[k])) || ((l!=0) && (arr[index]<arr[k]))) {
      temp = arr[index];
      arr[index] = arr[k];
      arr[k] = temp;
    }
  }
}


int cuda_sort(int number_of_elements, float *a)
{
  
  float *arr;
  
  hipMalloc((void**) &arr, number_of_elements * sizeof(float));
  hipMemcpy(arr, a, number_of_elements * sizeof(float), hipMemcpyHostToDevice);
  
  dim3 dimGrid(number_of_elements/THREADS);
  dim3 dimBlock(THREADS);

  for (int i = 2; i <= number_of_elements; i*=2) {
    int j = i/2;
    while (j > 0){
      bitonic_sort<<<dimGrid, dimBlock>>>(arr, i, j);
      j/=2;
    }
  }
  hipMemcpy(a, arr, number_of_elements * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(arr);

  return 0;
}

#ifdef __cplusplus
}
#endif

