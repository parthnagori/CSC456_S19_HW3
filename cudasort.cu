
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <sys/time.h>

#define THREADS 512
#ifdef __cplusplus
extern "C"
{
#endif

__global__ void
gpu_sort(int N, float *input, float *tmp, int x)
{
  // get the index of the current thread
  int y = blockIdx.x*blockDim.x + threadIdx.x;
  
  if (y < N && (y % (1 << (x+1)) == 0))
  {
    unsigned width = 1 << x;

    int left, middle, right;

  left = y;
  middle = y + width;
  right = y + 2*width;

  // merge function
  int i, j, k;
  i = left;
  j = middle;
  k = left;

  while(i < middle || j < right)
  {
    if (i < middle && j < right)
    {
      if (input[i] < input[j])
      {
        tmp[k++] = input[i++];
      }
      else
      {
        tmp[k++] = input[j++];
      }
    }
    else if (i == middle)
    {
      tmp[k++] = input[j++];
    }
    else if (j == right)
    {
      tmp[k++] = input[i++];
    }
  }

  // copy tmp back into input
  for(i = left; i < right; i++)
  {
    input[i] = tmp[i];
  }
  }
}

int cuda_sort(int number_of_elements, float *a)
{
  float *input_buf, *tmp_buf;

  // allocate device memory
  hipMalloc( (void **) &input_buf, sizeof(float) * number_of_elements );
  hipMalloc( (void **) &tmp_buf, sizeof(float) * number_of_elements);

  // move elements of a to the CUDA device
  hipMemcpy( input_buf, a, sizeof(float) * number_of_elements, hipMemcpyHostToDevice );

  unsigned blocks_per_grid = (number_of_elements + THREADS - 1) / THREADS;
  
  // determine what log2(N) is
  unsigned num_widths = 0;
  unsigned N = number_of_elements;
  while (N >>= 1) ++num_widths;

  // launch the kernel log2(N) times, each time setting a different
  // value for the 3rd kernel arguments. This value will be used to determine
  // the width of subarrays that the kernel should merge
  for (unsigned i = 0; i < num_widths; ++i)
  {
    gpu_sort<<<blocks_per_grid, THREADS>>>(number_of_elements, input_buf, tmp_buf, i);
  }
  
  hipMemcpy( a, input_buf, sizeof(float) * number_of_elements, hipMemcpyDeviceToHost);

  hipFree(input_buf);
  hipFree(tmp_buf);

  return 0;
}

#ifdef __cplusplus
}
#endif